#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 512

__global__ void matrixMultiply(int *A, int *B, int *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;  

    if (row < n && col < n) {
        int value = 0;
        for (int i = 0; i < n; i++) {
            value += A[row * n + i] * B[i * n + col];
        }
        C[row * n + col] = value;
    }
}

int main() {
    int *A, *B, *C;
    int *d_A, *d_B, *d_C;

    int size = N * N * sizeof(int);

    A = (int *)malloc(size);
    B = (int *)malloc(size);
    C = (int *)malloc(size);

    for (int i = 0; i < N * N; i++) {
        A[i] = i;
        B[i] = i * 2;
    }
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);
    matrixMultiply<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    printf("Matrix C (first 10 elements):\n");
    for (int i = 0; i < 10; i++) {
        printf("%d ", C[i]);
    }
    printf("\n");
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);

    return 0;
}
