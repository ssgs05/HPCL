#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 512

__global__ void matrixAdd(int *A, int *B, int *C, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        C[index] = A[index] + B[index];
    }
}

int main() {
    int *A, *B, *C;
    int *d_A, *d_B, *d_C;

    int size = N * N * sizeof(int);
    A = (int *)malloc(size);
    B = (int *)malloc(size);
    C = (int *)malloc(size);
    for (int i = 0; i < N * N; i++) {
        A[i] = i;
        B[i] = i * 2;
    }
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    int blockSize = 256;
    int numBlocks = (N * N + blockSize - 1) / blockSize;
    matrixAdd<<<numBlocks, blockSize>>>(d_A, d_B, d_C, N * N);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    printf("Matrix A + B (first 10 elements):\n");
    for (int i = 0; i < 10; i++) {
        printf("%d ", C[i]);
    }
    printf("\n");
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);

    return 0;
}
